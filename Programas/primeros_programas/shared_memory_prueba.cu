//Derivado de https://cuda-tutorial.readthedocs.io/ y
//https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
#define N 100000000

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int indice = blockIdx.x * blockDim.x + threadIdx.x;//Indice del thread que ejecuta el kernel
    int paso = blockDim.x * gridDim.x;//El numero de threads por bloque
    for(int i = indice; i < n; i+=paso){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; //Apuntadores a memoria compartida
    //Generamos los arreglos de memoria compartida
    hipMallocManaged(&a, sizeof(float) * N);
    hipMallocManaged(&b, sizeof(float) * N);
    hipMallocManaged(&out, sizeof(float) * N);
    // Inicializamos a y b
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    // Llamamos al kernel de CUDA (1 bloque, 256 threads por bloque).
    vector_add<<<100,256>>>(out, a, b, N);
    hipDeviceSynchronize();//Esperamos a que acaben todos los kernels.
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(out[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;
    hipFree(a);
    hipFree(b);
    hipFree(out);
    //Sugiero ver el comando: nvprof ./shared_memory_prueba
}
